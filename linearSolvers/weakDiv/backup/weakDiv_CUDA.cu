// ............................................................. //
//
// Function computing weak division A.x=B to obtain x in
// multiple cells, using CUDA. The vectors x and B can have
// multiple components (may not be fully supported yet).
//
// Manaure Francisquez.
// November 2019.
//
// ............................................................. //
#include <math.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)

int *solveLinearSystemsCUDA(const double *lhsA,const int probSize, double *rhsB,double *x,const int xDim,const int nProbs) {
  // Solve nProbs linear systems A.x=B of size probSize,
  // where x and B have xDim components.

  int *solveStatus;

  // Store the left-side matrix as an array of matrices.
  double **As = (double **)malloc(nProbs*sizeof(double *));
  for (int i=0; i<nProbs; i++) {

    int k = i*probSize;

    As[i] = (double*) calloc (probSize*probSize,sizeof(double));
    As[i][0] = 0.7071067811865475*lhsA[k+0];
    As[i][1] = 0.7071067811865475*lhsA[k+1];
    As[i][2] = 0.7071067811865475*lhsA[k+2];
    As[i][3] = 0.7071067811865475*lhsA[k+1];
    As[i][4] = 0.6324555320336759*lhsA[k+2]+0.7071067811865475*lhsA[k+0];
    As[i][5] = 0.6324555320336759*lhsA[k+1];
    As[i][6] = 0.7071067811865475*lhsA[k+2];
    As[i][7] = 0.6324555320336759*lhsA[k+1];
    As[i][8] = 0.4517539514526256*lhsA[k+2]+0.7071067811865475*lhsA[k+0];
  };

  // Store the right-side vector as an array of vectors.
  double **Bs = (double **)malloc(nProbs*sizeof(double *));
  for (int i=0; i<nProbs; i++) {
    Bs[i] = rhsB+i*probSize;
  };

  // Create CUBLAS solver.
  hipblasHandle_t cublasHandle = NULL;
  cublascall(hipblasCreate(&cublasHandle));

  // CUBLAS batched routines expect an array of pointers, each pointer
  // addressing a different linear problem. The procedure followed here is
  // to:
  //   1) Allocate a device-array of pointers, one entry per cell (A_d),
  //      and a device-array with the data of all cells (Aflat_d).
  //   2) Construct a host-array of pointers (A_h), each pointing to the address
  //      of the left-side matrix of the corresponding cell on the device.
  //   3) Copy the array of pointers (A_h) to the device-array of pointers (A_d).
  //   4) Copy the left-side matrix data (in As) to the device-array (Aflat_d),
  //      one cell at a time.
  double **A_d, *Aflat_d;
  cudacall(hipMalloc(&A_d,nProbs*sizeof(double *)));
  cudacall(hipMalloc(&Aflat_d, probSize*probSize*nProbs*sizeof(double)));
  double **A_h = (double **)malloc(nProbs*sizeof(double *));
  A_h[0] = Aflat_d;
  for (int i = 1; i < nProbs; i++)
    A_h[i] = A_h[i-1]+(probSize*probSize);
  cudacall(hipMemcpy(A_d,A_h,nProbs*sizeof(double *),hipMemcpyHostToDevice));
  for (int i = 0; i < nProbs; i++)
    cudacall(hipMemcpy(Aflat_d+(i*probSize*probSize), As[i], probSize*probSize*sizeof(double), hipMemcpyHostToDevice));

  // Perform the LU decomposition.
  const int lda = probSize;
  int *P_d;       // Pivots.
  int *info_d;    // Error info.
  cudacall(hipMalloc(&P_d, probSize * nProbs * sizeof(int)));
  cudacall(hipMalloc(&info_d, nProbs*sizeof(int)));
  cublascall(hipblasDgetrfBatched(cublasHandle,probSize,A_d,lda,P_d,info_d,nProbs));

  // Check that LU decomposition was successful.
  int infos[nProbs];
  cudacall(hipMemcpy(infos,info_d,nProbs*sizeof(int),hipMemcpyDeviceToHost));
  for (int i = 0; i < nProbs; i++)
    if (infos[i]  != 0)
    {
      fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
      hipDeviceReset();
      exit(EXIT_FAILURE);
    }

  // Allocate and assign the right-side device-vectors, following a procedure
  // similar to that in allocating and assigning the left-side device-matrices. 
  double **B_h = (double **)malloc(nProbs*sizeof(double *));
  double **B_d, *Bflat_d;
  cudacall(hipMalloc(&B_d,nProbs*sizeof(double *)));
  cudacall(hipMalloc(&Bflat_d, probSize*xDim*nProbs*sizeof(double)));
  B_h[0] = Bflat_d;
  for (int i = 1; i < nProbs; i++)
    B_h[i] = B_h[i-1] + probSize*xDim;
  cudacall(hipMemcpy(B_d,B_h,nProbs*sizeof(double *),hipMemcpyHostToDevice));
  for (int i = 0; i < nProbs; i++)
    cudacall(hipMemcpy(Bflat_d+(i*probSize*xDim), Bs[i], probSize*xDim*sizeof(double), hipMemcpyHostToDevice));

  // Compute the solution to the linear systems.
  int *info;    // Error info. NOTE: getrsBatched expects a host info, not a device info.
  const int ldb = probSize;
  cublascall(hipblasDgetrsBatched(cublasHandle,HIPBLAS_OP_N,probSize,xDim,(const double **)A_d,lda,P_d,B_d,ldb,info,nProbs));

//  // Copy solutions from device to host.
//  for (int i = 0; i < nProbs; i++)
//    cudacall(cudaMemcpy(x+(i*probSize), Bflat_d + (i*probSize), probSize*sizeof(double), cudaMemcpyDeviceToHost));

  // Free device memory.
  hipblasDestroy(cublasHandle);
  hipFree(A_d); hipFree(Aflat_d); hipFree(info_d);
  hipFree(P_d);
  hipFree(B_d); hipFree(Bflat_d);

  // Free host memory.
  free(As); free(Bs);
  free(A_h); free(B_h);

  solveStatus = 0;

  return solveStatus;
}


